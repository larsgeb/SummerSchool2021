
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void hello_kernel() {
    printf("hello world from cuda thread %d\n", int(threadIdx.x));
}

int main(void) {
    hello_kernel<<<1, 32>>>();
    //cudaDeviceSynchronize();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    double* ptr;

    auto size = 10 * sizeof(double);
    double *ptr_host = (double*)malloc(size);


    hipMalloc(&ptr, size);

    hipFree(ptr);

    cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    return 0;
}

